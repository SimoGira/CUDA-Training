/*
 * EXAMPLE OF TILED MATRIX-MATRIX MULTIPLICATION CHAPTER 4
 */
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

#define CHECK_ERROR(call) { \
	hipError_t err = call; \
	if (err != hipSuccess) { \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(err); \
	} \
}

#define TILE_WIDTH 4

__global__
void matrixMulKernel(float *P, float *M, float *N, int Width) {
    
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
    
    int tx = threadIdx.x, bx = blockIdx.x;
    int ty = threadIdx.y, by = blockIdx.y;
    
    // identify row and column of the d_P element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    
    if ( Row < Width && Col < Width ) {
        
        float pValue = 0;
        
        // Loop over the d_M and d_N tiles required to compute the d_P element
        for (int ph = 0; ph < Width/TILE_WIDTH; ph++) {
            
            // Collaborative loading of d_M and d_N tiles n to the shared memory
            Mds[ty][tx] = M[Row * Width + ph * TILE_WIDTH + tx];
            Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];
            
            __syncthreads();
            
            for(int k = 0; k < TILE_WIDTH; k++){
                pValue += Mds[ty][k]*Nds[k][tx];
            }
            __syncthreads();
        }
        P[Row*Width+Col] = pValue;
    }
}

void matrixMul(float *h_P, float *h_M, float *h_N, int dim) {
    
    int size = (dim*dim)*sizeof(float);
    float *d_M, *d_N, *d_P;
    
    //1. Allocate global memory on the device for d_Pin and d_Pout
    // With this type of allocation it isn't possible acces using higher-dimensional indexing syntax
    // it need to linearize first.
    CHECK_ERROR(hipMalloc((void**)&d_M, size));
    CHECK_ERROR(hipMalloc((void**)&d_N, size));
    CHECK_ERROR(hipMalloc((void**)&d_P, size));    // assume square matricies
    
    // copy h_Pin to device memory
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    
    //2. Kernel launch code - with TILe_WIDTH^2 threads per block
    dim3 dimGrid(ceil(dim / 4.0),ceil(dim / 4.0),1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH,1);
    matrixMulKernel<<<dimGrid, dimBlock>>>(d_P, d_M, d_N, dim);
    
    //3. copy d_Pout from the device memory
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    
    // Free device vectors
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

int main(int argc, char *argv[]) {
    
    float *h_M, *h_N, *h_P;
    int dim = 16; // assume square matricies
    
    h_M = (float*)malloc(sizeof(float)*dim*dim);
    h_N = (float*)malloc(sizeof(float)*dim*dim);
    h_P = (float*)malloc(sizeof(float)*dim*dim);
    
    // fill M and N with random float numbers
    srand(time(NULL));
    for (int i = 0; i < dim ; i++) {
        for (int j = 0; j < dim ; j++) {
            h_M[i*dim+j] = ((((float)rand() / (float)(RAND_MAX)) * 10));
            h_N[i*dim+j] = ((((float)rand() / (float)(RAND_MAX)) * 10));
        }
    }
    
    // perform matrix addiction
    matrixMul(h_P, h_M, h_N, dim);
    
    /*********************************************************************************************************
     // verifiy the result
     int valueIsCorrect = 1;
     float mult[dim][dim];
     
     for (int i = 0; i < dim; i++) {
     for (int j = 0; j < dim; j++) {
     mult[i][j] = 0.0;
     }
     }
     
     // Multiplying matrix firstMatrix and secondMatrix and storing in array mult.
     for(int i = 0; i < dim; ++i) {
     for(int j = 0; j < dim; ++j) {
     for(int k = 0; k < dim; ++k) {
     mult[i][j] += h_M[i*dim+k] * h_N[k*dim+j];
     }
     }
     }
     
     for (int i = 0; i < dim && valueIsCorrect; i++) {
     for (int j = 0; j < dim; j++) {
     printf("h_P[%d] != mult[%d][%d] --|-- %f != %f\n", (i*dim+j), i, j, h_P[i*dim+j], mult[i][j]);
     if (h_P[i*dim+j] != mult[i][j]) {
     valueIsCorrect = 0;
     printf("see error above.....\n");
     break;
     }
     }
     }
     ********************************************************************************************************
     * NON HA SENSO VERIFICARE LA CORRETTEZZA DEL RISULTATO SULL'HOST, VEDI 3.2 fino a 6.0 AL SEGUENTE LINK:
     * http://docs.nvidia.com/cuda/floating-point/
     ********************************************************************************************************/
    
    
    // Free host memory
    free(h_M);
    free(h_N);
    free(h_P);
    
    printf("ok multiplication completed with success!\n");
    
    /*
     if (valueIsCorrect) {
     printf("ok multiplication completed with success!\n");
     }
     else printf("somthing was wrong!\n");
     */
    
    return 0;
}
